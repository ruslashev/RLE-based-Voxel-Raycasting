#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
//#include <GL/glut.h>// Header File For The GLUT Library 

#define IN_CUDA_ENV

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  include <windows.h>
#endif
#include <cutil.h>
#include <cuda_gl_interop.h>
#include "cutil_math.h"
////////////////////////////////////////////////////////////////////////////////
#include "RayMap.h"
#include "Cuda_Render.h"
#include "Rle4.h"
// #include "../src.BestFitMem/bmalloc.h"
//#include "CudaMath.h"
////////////////////////////////////////////////////////////////////////////////

texture<uint2, 2, hipReadModeElementType> texture_pointermap;
texture<unsigned short, 1, hipReadModeElementType> texture_slabs;
#include "Cuda_Render.h"
////////////////////////////////////////////////////////////////////////////////

hipArray* cu_array;
hipChannelFormatDesc channelDesc;

void create_cuda_1d_texture(char* h_data, int size)
{
	int d_size = ((size >> 8)+1)<<8;
	printf("d_size %d size %d \n",d_size,size);
	uint *d_octree;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_octree, d_size));
    CUDA_SAFE_CALL(hipMemcpy((void *)d_octree, (void *)h_data, size, hipMemcpyHostToDevice) );
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);

    // set texture parameters
    texture_slabs.addressMode[0] = hipAddressModeClamp;
    texture_slabs.addressMode[1] = hipAddressModeClamp;
    texture_slabs.addressMode[2] = hipAddressModeClamp;
    texture_slabs.filterMode = hipFilterModePoint;
    texture_slabs.normalized = false;    // access with normalized texture coordinates
    CUDA_SAFE_CALL(hipBindTexture(0, texture_slabs, d_octree, channelDesc) );
}
////////////////////////////////////////////////////////////////////////////////

hipArray* cu_array_pointermap;
hipChannelFormatDesc channelDesc_pointermap;

void create_cuda_2d_texture(uint* h_data, int width,int height)
{
	// Allocate CUDA array in device memory 
    channelDesc_pointermap = 
               hipCreateChannelDesc(32, 32, 0, 0,	
			   hipChannelFormatKindUnsigned); 
	    
    hipMallocArray(&cu_array_pointermap, &channelDesc_pointermap, width, height); 
 
    // Copy to device memory some data located at address h_data 
    // in host memory  
    hipMemcpyToArray(cu_array_pointermap, 0, 0, h_data, width*height*8, 
                      hipMemcpyHostToDevice); 
 
    // Set texture parameters 
    texture_pointermap.addressMode[0] = hipAddressModeClamp; 
    texture_pointermap.addressMode[1] = hipAddressModeClamp; 
    texture_pointermap.addressMode[2] = hipAddressModeClamp;
    texture_pointermap.filterMode     = hipFilterModePoint; 
    texture_pointermap.normalized     = false; 
 
    // Bind the array to the texture 
    hipBindTextureToArray(
		texture_pointermap, 
		cu_array_pointermap, 
		channelDesc_pointermap); 
	
	/*
	int d_size = (((size >> 8)+1)<<8);
	printf("d_size %d size %d \n",d_size,size);
	uint *d_data;

	channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);//<unsigned int>();//<unsigned int>();//
	CUDA_SAFE_CALL( hipMallocArray( &cu_array, &channelDesc, tex_w, tex_h )); 
	CUDA_SAFE_CALL( hipMemcpyToArray( cu_array, 0, 0, (void*)(texdata) , tex_w*tex_h*4, hipMemcpyHostToDevice));

	texture_array.addressMode[0] = hipAddressModeWrap;
	texture_array.addressMode[1] = hipAddressModeWrap;
    texture_array.addressMode[2] = hipAddressModeClamp;
	texture_array.filterMode = hipFilterModePoint;//hipFilterModeLinear;
	texture_array.normalized = false;    // access with normalized texture coordinates

	// Bind the array to the texture
	CUDA_SAFE_CALL( hipBindTextureToArray( texture_array, cu_array, channelDesc));
    CUDA_SAFE_CALL( hipDeviceSynchronize() );



    CUDA_SAFE_CALL(hipMalloc((void**) &d_octree, d_size));
    CUDA_SAFE_CALL(hipMemcpy((void *)d_octree, (void *)h_data, size, hipMemcpyHostToDevice) );
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

    // set texture parameters
    texture_array.addressMode[0] = hipAddressModeClamp;
    texture_array.addressMode[1] = hipAddressModeClamp;
    texture_array.filterMode = hipFilterModePoint;
    texture_array.normalized = false;    // access with normalized texture coordinates
    CUDA_SAFE_CALL(hipBindTexture(0, texture_array, d_octree, channelDesc) );
	*/
}
////////////////////////////////////////////////////////////////////////////////
// GL ERROR CHECK
int ChkGLError(char *file, int line)
{
	//return 0;
	return 0;
}
#define C_CHECK_GL_ERROR() ChkGLError(__FILE__, __LINE__)
////////////////////////////////////////////////////////////////////////////////
extern "C" void cuda_main_render2( int pbo_out, int width, int height,RayMap_GPU* raymap);
extern "C" void pboRegister(int pbo);
extern "C" void pboUnregister(int pbo);
int	cpu_to_gpu_delta=0;
////////////////////////////////////////////////////////////////////////////////
void gpu_memcpy(void* dst, void* src, int size)
{
	CUDA_SAFE_CALL( hipMemcpy( dst, src, size, hipMemcpyHostToDevice) );
	CUT_CHECK_ERROR("hipMemcpy hipMemcpyHostToDevice failed");
}
////////////////////////////////////////////////////////////////////////////////
void cpu_memcpy(void* dst, void* src, int size)
{
	CUDA_SAFE_CALL( hipMemcpy( dst, src, size, hipMemcpyDeviceToHost) );
	CUT_CHECK_ERROR("hipMemcpy hipMemcpyDeviceToHost failed");
}
////////////////////////////////////////////////////////////////////////////////
void* gpu_malloc(int size)
{
	void* ptr=0;	
	CUDA_SAFE_CALL( hipMalloc( (void**) &ptr, size ) );
	CUT_CHECK_ERROR("hipMalloc failed");
	if(ptr==0){printf("\ncudaMalloc %d MB: out of memory error\n",(size>>20));while(1);;}
	return ptr;
}
////////////////////////////////////////////////////////////////////////////////
__global__ void
cudaRender(
		   Render* render_local,
		   int maxrays, 
		   vec3f viewpos, 
		   vec3f viewrot, 
		   int res_x, 
		   int res_y,
		   ushort* skipmap_gpu
		  )
{
    extern __shared__ int sdata[];
   
    int x = ( blockIdx.y * 2 + blockIdx.x )* blockDim.x + threadIdx.x;
   
	//if(x&1)return;
    if (x>=maxrays) return;
    
    //render_local->render_line(x,(unsigned int*)&sdata[((x)&127)*31]);
    render_local->render_line
	(
		x,
		(unsigned int*)&sdata[((x)&(THREAD_COUNT-1))*(16300/(THREAD_COUNT*4))],//31
		viewpos,
		viewrot,
		res_x,
		res_y,
		skipmap_gpu+x*res_y
	);

	return;
}
////////////////////////////////////////////////////////////////////////////////
void cuda_main_render2( int pbo_out, int width, int height,RayMap_GPU* raymap)
{
	// int t0 = timeGetTime();

	if(pbo_out==0) return;

    static Render render;
    static Render *render_gpu=(Render*) ((char*)malloc(sizeof(Render))+cpu_to_gpu_delta);
    static ushort* skipmap_gpu=(ushort*)((char*)malloc(RAYS_CASTED*RENDER_SIZE*4)+cpu_to_gpu_delta);
    
    if((long)render_gpu==cpu_to_gpu_delta){ printf("render_gpu 0 \n");while(1);;}
    int lines_to_raycast = raymap->map_line_count;
    int thread_calls = ((raymap->map_line_count/2) | (THREAD_COUNT-1)) +1;
    if (lines_to_raycast>RAYS_CASTED ) lines_to_raycast=RAYS_CASTED;
    int* out_data;   
    CUDA_SAFE_CALL(cudaGLMapBufferObject( (void**)&out_data, pbo_out));   
	if(out_data==0) return;

	dim3 threads(THREAD_COUNT,1,1 );
    dim3 grid( 2 , thread_calls /(threads.x),1 );

    render.set_target( width, height, (int*) out_data);
  	render.set_raymap( raymap );

#ifdef DETAIL_BENCH
	for(int t=0;t<RAYS_CASTED;t++)
	{
		render.perf[t].elems_total=0;
		render.perf[t].elems_processed=0;
		render.perf[t].voxels_processed=0;
		render.perf[t].elems_rendered=0;
		render.perf[t].pixels=0;
	}
#endif
	
	gpu_memcpy(render_gpu, &render, sizeof(Render));
   
	// int t1 = timeGetTime();
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	//printf("before\n");
	//Sleep(10000);

	if(1)
	cudaRender<<< grid, threads, 16300 >>>
	(
		render_gpu,
		render.ray_map.map_line_count,
		render.ray_map.position,
		render.ray_map.rotation,
		render.res_x,
		render.res_y,
		skipmap_gpu
	);
	
	CUT_CHECK_ERROR("cudaRender failed");
//	CUT_CHECK_ERROR_GL();
	C_CHECK_GL_ERROR();

	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	// int t2 = timeGetTime();

#ifdef DETAIL_BENCH
	cpu_memcpy(&render.perf[0],&(render_gpu->perf[0]),  sizeof(Render::Perf)*RAYS_CASTED);
	Render::Perf p;
	p.elems_total=0;
	p.elems_processed=0;
	p.voxels_processed=0;
	p.elems_rendered=0;
	p.pixels=0;
	for(int t=0;t<RAYS_CASTED;t++)
	{
		p.elems_total+=render.perf[t].elems_total;
		p.elems_processed+=render.perf[t].elems_processed;
		p.voxels_processed+=render.perf[t].voxels_processed;
		p.elems_rendered+=render.perf[t].elems_rendered;
		p.pixels+=render.perf[t].pixels;
	}
	
	printf ("all %2.2fM proc %2.2fM vp %2.2fM ren %2.2fM pix %2.2fM ",
		float(p.elems_total)/(1000*1000),
		float(p.elems_processed)/(1000*1000),
		float(p.voxels_processed)/(1000*1000),
		float(p.elems_rendered)/(1000*1000),
		float(p.pixels)/(1000*1000));
#endif		
	//printf ("mem%d ren%d ",t1-t0,t2-t1);
    
    CUDA_SAFE_CALL(cudaGLUnmapBufferObject( pbo_out));
}
////////////////////////////////////////////////////////////////////////////////
void pboRegister(int pbo)
{
    // register this buffer object with CUDA
    CUDA_SAFE_CALL(cudaGLRegisterBufferObject(pbo));
	CUT_CHECK_ERROR("cudaGLRegisterBufferObject failed");
	C_CHECK_GL_ERROR();
}
////////////////////////////////////////////////////////////////////////////////
void pboUnregister(int pbo)
{
    // unregister this buffer object with CUDA
    CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(pbo));	
	CUT_CHECK_ERROR("cudaGLUnregisterBufferObject failed");
	C_CHECK_GL_ERROR();
}
////////////////////////////////////////////////////////////////////////////////
/*
__global__ void
cudaColorNodes(uint* nodebuf)
{
    int x = (blockIdx.x * blockDim.x + threadIdx.x);
    int y = (blockIdx.y * blockDim.y + threadIdx.y);

	ushort* node = (ushort*)(((uint*)nodebuf) [x+y*1024]);

	uint col_rgb=0xff8844;
	if(node)
	{
		ushort col=(ushort)*node;

		const int col_r[4]={130 ,255,255,155};
		const int col_g[4]={255 ,155,0  ,255};
		const int col_b[4]={130 ,0  ,0  ,0};						

		int col_o=(col>>8)&3;				
		int bright = col&255 ;

		int r_=(bright*col_r[col_o])>>8;
		int g_=(bright*col_g[col_o])>>8;
		int b_=(bright*col_b[col_o])>>8;

		col_rgb = r_+(g_<<8)+(b_<<16) ;
	}

	((uint*)nodebuf) [x+y*1024] = col_rgb;
}
*/
////////////////////////////////////////////////////////////////////////////////
