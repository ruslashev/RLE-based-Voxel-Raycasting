#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
//#include <GL/glut.h>// Header File For The GLUT Library 

#define IN_CUDA_ENV

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  include <windows.h>
#endif
#include <cutil.h>
#include <cuda_gl_interop.h>
#include "cutil_math.h"
#include "mathlib/matrixdefs.h"

////////////////////////////////////////////////////////////////////////////////
#include "alloc.hh"
#include "RayMap.hh"
#include "Rle4.hh"
// #include "../src.BestFitMem/bmalloc.h"
//#include "CudaMath.h"
////////////////////////////////////////////////////////////////////////////////

texture<uint2, 2, hipReadModeElementType> texture_pointermap;
texture<unsigned short, 1, hipReadModeElementType> texture_slabs;

////////////////////////////////////////////////////////////////////////////////

struct Render
{
    /*-------------------- Variables    --------------------*/

	RayMap_GPU ray_map;

	int res_x,res_y;
	int *data_rgb;

	struct Perf
	{
		int elems_total;
		int elems_processed;
		int voxels_processed;
		int elems_rendered;
		int pixels;
	};
	Perf perf[RAYS_CASTED];


    /*------------------------------------------------------*/
	void set_target(  int resolution_x, int resolution_y, int* data_rgb )
	{
		this->res_x=resolution_x;
		this->res_y=resolution_y;
		this->data_rgb=data_rgb;
		if (data_rgb==0) {printf("data_rgb 0\n");while(1);}
	}
	/*------------------------------------------------------*/
	void set_raymap( RayMap_GPU* raymap )
	{
		memcpy (&ray_map,raymap,sizeof(RayMap_GPU));
	}
	/*------------------------------------------------------*/
	inline __device__ float LineScale(vec3f input, vec3f center,float clip_max,float clip_min)
	{
		float scale_x = 1;
		float scale_y = 1;

		if (center.x>1)	scale_x = (1-input.x)/(center.x-input.x);
		if (center.x<0)	scale_x = input.x/(input.x-center.x);
		if (center.y>clip_max)	scale_y = ( clip_max-input.y)/(center.y-input.y);
		if (center.y<clip_min)	scale_y = (-clip_min+input.y)/(input.y-center.y);

		float scale = (scale_x<scale_y) ? scale_x : scale_y;

		return scale;
	}
	/*------------------------------------------------------*/
	inline __device__ void ClipLine(vec3f &p1, vec3f &p2,float clip_max,float clip_min)
	{
		vec3f c1,c2; float scale; 
		
		c1=p1;c2=p2;

		scale = LineScale(p1,p2,clip_max,clip_min);	c2 = p1+(p2-p1) * scale;
		scale = LineScale(p2,p1,clip_max,clip_min);	c1 = p2+(p1-p2) * scale;

		p1 = c1;
		p2 = c2;
	}
	/*------------------------------------------------------*/
	inline __device__ vec3f MatMul ( matrix44 m, vec3f v)
	{
		return make_float3(
		m.M11*v.x + m.M21*v.y + m.M31*v.z + m.M41,
		m.M12*v.x + m.M22*v.y + m.M32*v.z + m.M42,
		m.M13*v.x + m.M23*v.y + m.M33*v.z + m.M43);
	}
	/*------------------------------------------------------*/
    inline __device__ void vec3f_rot_y( float a , vec3f& v) 
    { 
		float xx = cos ( -a ) * v.x + sin ( -a ) * v.z;
		float zz = cos ( -a ) * v.z - sin ( -a ) * v.x;
		v.x=xx;v.z=zz;
	}

	/*------------------------------------------------------*/
    inline __device__ void vec3f_normalize( vec3f& v )
    { 
		float square = v.x*v.x + v.y*v.y + v.z*v.z;
		if (square <= 0.00001f ) 
		{
			v.x=1;v.y=0;v.z=0;
			return; 
		}
		float len = 1.0 / (float)sqrt(square); 
		v.x*=len;v.y*=len;v.z*=len;
		return; 
	}
	/*------------------------------------------------------*/
	inline __device__ void render_line 
	(
		int x, 
		unsigned int *y_cache, 
		vec3f viewpos ,
		vec3f viewrot ,
		int res_x,
		int res_y,
		ushort* ofs_skip_start
	)
	{
		float res_x2 = res_x/2;
		float res_y2 = res_y/2;
		uint*  ofs_rgb_start = (uint*)&data_rgb[x*res_y];
		uint*  ofs_cache_start = ((uint*)ofs_skip_start)+x*res_y;

		ofs_cache_start[0]=0;
		uint ofs_cache_count = 0;
		uint ofs_cache_depth = 0;

		//RayMap_GPU::MapLine ml = ray_map.map_line[x];

		float ml_ray_x;//=ml.ray_x;
		float ml_ray_z;//=ml.ray_z;
		vec3f ml_start2d;//=ml.start2d;
		vec3f ml_end2d;//=ml.end2d;
		vec3f ml_start3d;//=ml.start3d;
		vec3f ml_end3d;//=ml.end3d;
		bool  ml_direction_y;//=ml.direction_y;

		//if(0)
		//if(x>=ray_map.res[0]/2)
		{
			int rays[4];
			rays[0]=ray_map.res[0];
			rays[1]=ray_map.res[1]+rays[0];
			rays[2]=ray_map.res[2]+rays[1];
			rays[3]=ray_map.res[3]+rays[2];

			int quadrant=0;
			if (x>=rays[2]) quadrant=3;else
			if (x>=rays[1]) quadrant=2;else
			if (x>=rays[0]) quadrant=1;
			
			float quadrant_ofs = x;
			if(quadrant>=1) quadrant_ofs -= rays[quadrant-1];

			float quadrant_num = ray_map.res[quadrant];
			float a = quadrant_ofs / quadrant_num;

			int j=quadrant;

			vec3f p1,p2,p1_3d,p2_3d;
			p1 = ray_map.p_2d[5];
			p2 = ray_map.p_no[j*2]+(ray_map.p_no[j*2+1]-ray_map.p_no[j*2])*a;
			
			ClipLine(p1,p2,ray_map.clip_max,ray_map.clip_min);

			matrix44 to3d = ray_map.to3d;
			vec3f p1m4 = p1*4.0;
			vec3f p2m4 = p2*4.0;
			p1_3d = MatMul ( to3d , p1m4 );
			p2_3d = MatMul ( to3d , p2m4 );

			vec3f delta = (p1_3d+p2_3d)*0.5-ray_map.p4;
			delta.y = 0;
			delta = normalize(delta);
			vec3f_rot_y( viewrot.y , delta);

			ml_ray_x = delta.x;				
			ml_ray_z = delta.z;			
			ml_start2d=p1;
			ml_end2d=p2;
			ml_start3d=p1_3d;
			ml_end3d=p2_3d;
			ml_direction_y = 1-(j>>1);
		}

		/////////////////////////////////////////// 
		// Initialize Render Vars

		int mip_lvl=0;
		int y_clip_min	= 0;
		int y_clip_max	= res_y-1;
		const int z_far = RAYS_DISTANCE;
		float dz		= 1<<mip_lvl;
		int mapswitch	= MIP_DISTANCE;//res_y2 	;
		//if (SCREEN_SIZE_X < res_y) mapswitch	= SCREEN_SIZE_X;

		/////////////////////////////////////////// 
		// Initialize Rotation Vars

		float sin_x = sin(ray_map.rotation.x); // Rotation around x-axis
		float cos_x = cos(ray_map.rotation.x);
		float sin_y = sin(ray_map.rotation.y); // Rotation around y-axis
		float cos_y = cos(ray_map.rotation.y);

		/////////////////////////////////////////// 
		// Initialize Raymap Vars

		float ray_x = ml_ray_x;
		float ray_z = ml_ray_z;
		bool vertical = ml_direction_y;

		/////////////////////////////////////////// 
		// Reverse texturing ?

		bool reverse=false;
		if( vertical)if(ray_z<=0) reverse=true;
		if(!vertical)if(ray_x<=0)if(sin_x>0) reverse=true;
		if(!vertical)if(ray_x> 0)if(sin_x<0) reverse=true;

		float res_x2_mul_reverse   = (reverse) ? -res_x2 : res_x2;

		if(vertical)	res_x2_mul_reverse = -res_x2_mul_reverse; 

		/////////////////////////////////////////// 
		// Screenspace clipping

		int3 p1,p2;
		int p_add=(reverse)?1:-2;
		p1.x = int(float(float(res_x) * ml_start2d.x))+p_add;
		p1.y = int(float(float(res_y) * ml_start2d.y))+p_add;
		p2.x = int(float(float(res_x) * ml_end2d.x))-p_add;
		p2.y = int(float(float(res_y) * ml_end2d.y))-p_add;
		if (p1.x<0) p1.x = 0;if (p1.x>=res_x) p1.x = res_x-1;
		if (p1.y<0) p1.y = 0;if (p1.y>=res_y) p1.y = res_y-1;
		if (p2.x<0) p2.x = 0;if (p2.x>=res_x) p2.x = res_x-1;
		if (p2.y<0) p2.y = 0;if (p2.y>=res_y) p2.y = res_y-1;

		if(p1.y==p2.y) return; // If removed -> Error ..!!?? Todo

		y_clip_min = res_x-1-p1.x;
		y_clip_max = res_x-1-p2.x;

		if(vertical)
		{
			y_clip_min = res_y-1-p1.y;
			y_clip_max = res_y-1-p2.y;
		}

		if(reverse)
		{
			y_clip_min = res_y-1-y_clip_min;
			y_clip_max = res_y-1-y_clip_max;
		}

		if (y_clip_min > y_clip_max)
		{
			int tmp = y_clip_min;
			y_clip_min = y_clip_max;
			y_clip_max = tmp;
		}

		if (y_clip_min >= y_clip_max) return;

		/////////////////////////////////////////// 
		// Clear current rendered Line

		for (int n=y_clip_min;n<=y_clip_max;n++)
		{
			ofs_rgb_start[n]=0xff8844;//400000.0;//x123456*n;
			#ifdef PERPIXELFORWARD
			 ofs_skip_start[n]=0;//400000.0;//x123456*n;
			#endif			 
		}
#ifdef SHAREMEMCLIP
		for (int n=0;n<31;n++) y_cache[n]=0;
#endif			 

		/////////////////////////////////////////// 
		//
		//void RayDDS(vec3f direction,MapLine& ml)
		
		float2 direction_rot;
		direction_rot.x = ray_x * cos_y + ray_z * sin_y;
		direction_rot.y = ray_x * sin_y - ray_z * cos_y;

		float2 delta = direction_rot, frac, fix, sign;

		fix.x  = fix.y  =-1;
		frac.x = viewpos.x - int(viewpos.x);
		frac.y = viewpos.z - int(viewpos.z);
		sign.x = sign.y = -1;

		// Signs & direction for frac
		if (delta.x >= 0) { fix.x =0; sign.x = 1; frac.x = 1-frac.x; }
		if (delta.y >= 0) { fix.y =0; sign.y = 1; frac.y = 1-frac.y; }

		// Gradients    
		float2 grad0,grad1;
		grad0.y = delta.y / fabs(delta.x); grad0.x = sign.x;
		grad1.x = delta.x / fabs(delta.y); grad1.y = sign.y;

		// Intersections in x-,y- and z-plane
		float2 isect0,isect1; 
		isect0.x = grad0.x * frac.x;
		isect0.y = grad0.y * frac.x;
		isect1.x = grad1.x * frac.y;
		isect1.y = grad1.y * frac.y;

		float grad_dist0 = sqrt(grad0.x*grad0.x+grad0.y*grad0.y);
		float grad_dist1 = sqrt(grad1.x*grad1.x+grad1.y*grad1.y);
		float dds_dist0= sqrt(isect0.x*isect0.x+isect0.y*isect0.y);
		float dds_dist1= sqrt(isect1.x*isect1.x+isect1.y*isect1.y);

		float2 pos_vxl_before= {0,0}; float dds_dist_before =0;
		float2 pos_vxl		 = {0,0}; float dds_dist_now    =0; 

		int index=0,index_before=0;
		/////////////////////////////////////////// 
		//
		// Main Render Loop

		int	rle4_gridx=ray_map.map4_gpu[mip_lvl].sx;
		int	rle4_gridz=ray_map.map4_gpu[mip_lvl].sz;

		float pos3d_z_add = sin_x;
		float pos3d_y_add = (vertical) ? cos_x:0;
		  	  pos3d_y_add*=  res_x2_mul_reverse;

#ifdef DETAIL_BENCH
		int numpix = 0;
		int rndpix = y_clip_max-y_clip_min;
#endif

		uint*    map_ptr = ray_map.map4_gpu[mip_lvl].map;
		ushort* slab_ptr = ray_map.map4_gpu[mip_lvl].slabs;

		float z=0;

#ifdef CENTERSEG
		float cache_1_start	= y_clip_max;
		float cache_1_end	= y_clip_min;
#endif
		bool skipme=false;

		float y_map_switch = viewpos.y;

		mapswitch = mapswitch * (0.25*(4-abs(viewrot.x)));

		uint tex_map_ofs=0;
		
while(true){

			ofs_cache_depth++;

			while ( z>mapswitch || (y_map_switch>512.0)){

			y_map_switch = y_map_switch * 0.5;

			if (mip_lvl<ray_map.nummaps-1)
			{
				mip_lvl++;
				tex_map_ofs+=rle4_gridz;//tex_map_add;
				rle4_gridx>>=1;
				rle4_gridz>>=1;
				map_ptr = ray_map.map4_gpu[mip_lvl].map;
				slab_ptr= ray_map.map4_gpu[mip_lvl].slabs;
			}
			
			grad0.x *= 2;
			grad0.y *= 2;
			grad1.x *= 2;
			grad1.y *= 2;
			grad_dist0 *=2;
			grad_dist1 *=2;
			mapswitch *= 2;
			dz*=2;
		}
		z+=dz;
		if (z>z_far)return;

#ifndef DETAIL_BENCH
		if ((y_clip_min>=y_clip_max))	return;
//		if ((y_clip_min>>1>=y_clip_max>>1)||(numpix==rndpix))	return;
#endif

		/////////////////////////////////////////// 
		//DDA				
		dds_dist_before	= dds_dist_now;
		pos_vxl_before		= pos_vxl;
		index_before		= index;

		//dds_dist0= sqrt(isect0.x*isect0.x+isect0.y*isect0.y);
		//dds_dist1= sqrt(isect1.x*isect1.x+isect1.y*isect1.y);

		/*
		float if1 = ( dds_dist1 < dds_dist0) ? 1.0 : 0.0;
		float if0 = 1.0-if1;
		dds_dist_now = dds_dist1*if1 + dds_dist0*if0;
		index = if1;
		pos_vxl.x	  = isect1.x*if1+isect0.x*if0;
		pos_vxl.y	  = isect1.y*if1+isect0.y*if0;
		dds_dist1+=grad_dist1*if1;
		isect1.x += grad1.x*if1;
		isect1.y += grad1.y*if1;
		dds_dist0+=grad_dist0*if0;
		isect0.x += grad0.x*if0;
		isect0.y += grad0.y*if0;
		*/
		
		if ( dds_dist1 < dds_dist0)	
		{ 
			dds_dist_now = dds_dist1;
			index = 1; 
			dds_dist1+=grad_dist1;//dds_dist1; 
			pos_vxl	  = isect1  ;
			isect1.x += grad1.x ;
			isect1.y += grad1.y ;
		}else
		{
			dds_dist_now = dds_dist0;
			index = 0;
			pos_vxl	  = isect0  ;
			dds_dist0+=grad_dist0;//dds_dist1; 
			isect0.x += grad0.x ;
			isect0.y += grad0.y ;
		}

		if(skipme){skipme=false;continue;}

		int fix_x=(1-index_before)*fix.x,
			fix_z=(  index_before)*fix.y;

		/////////////////////////////////////////// 

		float dds_dist_delta = dds_dist_now-dds_dist_before;

		vec3f view_space;
		view_space.x = ray_x * dds_dist_before;
		view_space.z = ray_z * dds_dist_before;

		int voxel_x = (int( viewpos.x+pos_vxl_before.x )+fix_x);
		int voxel_z = (int( viewpos.z+pos_vxl_before.y )+fix_z);

#ifdef CLIPREGION
		if(voxel_x<0)continue;
		if(voxel_z<0)continue;
		if((voxel_x>>mip_lvl)>rle4_gridx-1)continue;
		if((voxel_z>>mip_lvl)>rle4_gridz-1)continue;
#endif				
		//if (voxel_x&1024) continue;
		//if (voxel_z&1024) continue;

		int vx = (voxel_x>>mip_lvl) & (rle4_gridx-1);
		int vz = (voxel_z>>mip_lvl) & (rle4_gridz-1);

#ifdef MOUNTAINS
		float xx1=float(voxel_x)/1000.0f;
		float zz1=float(voxel_z)/1000.0f;
//		float xx1=float(voxel_x)/1000.0f;
//		float zz1=float(voxel_z)/1000.0f;
		float sinadd =(sin(xx1/2+zz1/3)+cos(zz1/2+xx1)*0.5+cos(zz1))*500+500;
		float sinadd2=(sin(xx1/17+zz1/19)+cos(zz1/12+xx1/13)*0.5+cos(zz1/31))*4500+4500;
		float mountain = viewpos.y+sinadd;//+sinadd2;
#else
		float mountain = viewpos.y;
#endif

		float correct_x = ray_x * dds_dist_delta;
		float correct_z = ray_z * dds_dist_delta;

		float pos3d_z = cos_x*(view_space.z) + sin_x*mountain;
		float pos3d_y = (vertical) ? 
			  cos_x* mountain - sin_x*(view_space.z)
			: view_space.x;

		pos3d_y	*= res_x2_mul_reverse;

		// Clip Top //if (pos3d_z>0) 
		if (pos3d_z*res_y2 + pos3d_y <=pos3d_z*y_clip_min) continue;

#ifdef MEM_TEXTURE
		uint2 int64_ofs_rle=tex2D(
			texture_pointermap, 
			vx,vz+tex_map_ofs);
#else
		uint2 int64_ofs_rle=((uint2*)map_ptr)[vx+vz*rle4_gridx];
#endif
		uint slab_offset= int64_ofs_rle.x;
		uint len_first  = int64_ofs_rle.y; 
		ushort  slen = len_first; //if(slen==0)continue;

		float pos3d_z1,pos3d_z2;int scr_y1;
		float pos3d_y1,pos3d_y2;int scr_y2;
		
		float corr_zz= cos_x*correct_z;
		float corr_yy= (vertical) ?  -sin_x*correct_z : correct_x;

		corr_yy *= res_x2_mul_reverse;

		int sti_general = 0 ; 
		int sti_skip = 0;
		ushort sti_ ;

		sti_ = len_first >> 16;

#ifdef MEM_TEXTURE
		uint slabs = 2+slab_offset;
		uint send = slabs + slen;
#else
		ushort* slabs = slab_ptr+2+slab_offset;
		ushort* send = slabs + slen;
#endif
		float tex	 = 0;
		sti_general = 0 ; 
		sti_skip = 0;

#ifdef DETAIL_BENCH
		perf[x].elems_total+=slen;
		if ((y_clip_min>>1>=y_clip_max>>1)||(numpix==rndpix))	continue;
#endif
		uint slabs1=(uint)slabs;

		#pragma unroll 2
		for (;slabs<send;++slabs)
		{			
#ifdef MEM_TEXTURE
			if ( uint(slabs) > slabs1 )	sti_ = tex1Dfetch(texture_slabs, slabs);
#else
			if ( uint(slabs) > slabs1 )	sti_ = *slabs;
#endif

 
			sti_skip     = (sti_>>10 );	
			sti_general += (sti_&1023)<<mip_lvl;
			if (sti_skip==0)continue;

			int texture=tex;
			tex+=sti_skip;
			sti_skip <<= mip_lvl;

			float sti_general_sti_skip=sti_general;
			sti_general+=sti_skip;
		
			float correct_zz1=pos3d_z;
			float correct_yy1=pos3d_y;
			if ( mountain+sti_general_sti_skip>=0){correct_zz1+=corr_zz;correct_yy1+=corr_yy;} 

#ifdef DETAIL_BENCH
		perf[x].voxels_processed+=sti_skip;
		perf[x].elems_processed++;
#endif
			pos3d_z1=correct_zz1+pos3d_z_add*sti_general_sti_skip;if (pos3d_z1<=0) continue;
			pos3d_y1=correct_yy1+pos3d_y_add*sti_general_sti_skip;
			scr_y2 = res_y2 + pos3d_y1 / pos3d_z1 ;			
			if (scr_y2<=y_clip_min){ skipme=false;break; }

			/*
			if ( uint(slabs) == slabs1 ) // column visible
			{
				ofs_cache_count ++;
				ofs_cache_start[0]=ofs_cache_count;
				ofs_cache_start[ofs_cache_count]=ofs_cache_depth;
			}
			*/

			float correct_zz2=pos3d_z;
			float correct_yy2=pos3d_y;
			if ( mountain+sti_general <0){correct_zz2+=corr_zz;correct_yy2+=corr_yy;}

			pos3d_z2=correct_zz2+pos3d_z_add*sti_general;if (pos3d_z2<=0) continue;
			pos3d_y2=correct_yy2+pos3d_y_add*sti_general;
			scr_y1 = res_y2 + pos3d_y2 / pos3d_z2-1;if (scr_y1>=y_clip_max)	continue;
			
			
#ifndef CENTERSEG
			if (scr_y2>=y_clip_max){ 
				scr_y2=y_clip_max;
				#ifdef FLOATING_HORIZON
				y_clip_max = scr_y1;
				#endif
			}
			if (scr_y1<=y_clip_min){ 
				scr_y1=y_clip_min;
				#ifdef FLOATING_HORIZON
				y_clip_min = scr_y2;
				#endif
				#ifdef SHAREMEMCLIP
				#ifdef XFLOATING_HORIZON
				while( (y_clip_max>y_clip_min) && (y_cache[y_clip_min>>5]&(1<<(y_clip_min&31))) )++y_clip_min;
				#endif		
				#endif
			}
#else
			bool merged = false;

			if (scr_y1<y_clip_min){ if ( y_clip_min >= scr_y2 ) continue; scr_y1=y_clip_min;y_clip_min = scr_y2; merged =true; }
			if (scr_y2>y_clip_max){ if ( scr_y1 >= y_clip_max ) continue; scr_y2=y_clip_max;y_clip_max = scr_y1; merged =true; }

			/////////////////////////////////////////// 
			// Culling #2 - Check center segment

			bool y1gc1s = scr_y1>=cache_1_start;
			bool y2lc1e = scr_y2<=cache_1_end;
			
			if (y1gc1s && y2lc1e) continue;

			if (!merged) if (!y1gc1s || !y2lc1e)	
			{
				if (scr_y1<cache_1_start)
				if (scr_y2>=cache_1_start)
				if (y2lc1e)
				{
					scr_y2 = cache_1_start;
					cache_1_start=scr_y1;
				}
				if (scr_y2>cache_1_end)
				if (scr_y1<=cache_1_end)
				if (y1gc1s)
				{
					scr_y1 = cache_1_end;
					cache_1_end=scr_y2;
				}
				if (scr_y2-scr_y1 > cache_1_end-cache_1_start)
				{
					cache_1_start = scr_y1;
					cache_1_end = scr_y2;
				}
			}
#endif		
			/////////////////////////////////////////// 
			int y=scr_y1;

#ifdef PERPIXELFORWARD
	        y     +=ofs_skip_start[y];//&65535;
			//scr_y2-=ofs_skip_start[scr_y2-1]>>16;
#ifdef XFLOATING_HORIZON
			y_clip_min+=ofs_skip_start[y_clip_min];//&65535;
#endif
			if (y>=scr_y2) continue;
#endif
#ifdef NORMALCLIP
//			while( (y<scr_y2) && (((uint*)ofs_rgb_start)[y]!=0xff8844) ) ++y;
//			while( (y_clip_max>y_clip_min) && (((uint*)ofs_rgb_start)[y_clip_min]!=0xff8844)  )++y_clip_min;
#endif
#ifdef SHAREMEMCLIP
#ifdef CENTERSEG
#ifdef XFLOATING_HORIZON
			while( (y_clip_max>y_clip_min) && (y_cache[y_clip_min>>5]&(1<<(y_clip_min&31))) )++y_clip_min;
#endif
#endif
			while( (y<scr_y2) && (y_cache[y>>5]&(1<<(y&31))) )++y;
			if (y>=scr_y2) continue;
#endif
			float scr_y1r , pos3d_z2r , pos3d_y2r ;
			float scr_y2r , pos3d_z1r , pos3d_y1r ;

			pos3d_z1r=pos3d_z+pos3d_z_add*sti_general_sti_skip;
			pos3d_y1r=pos3d_y+pos3d_y_add*sti_general_sti_skip;

			pos3d_z2r=pos3d_z+pos3d_z_add*sti_general;
			pos3d_y2r=pos3d_y+pos3d_y_add*sti_general;

			scr_y2r = res_y2 + pos3d_y1r / pos3d_z1r ;			
			scr_y1r = res_y2 + pos3d_y2r / pos3d_z2r ;			

			float u1z = (tex    ) / pos3d_z2r;     
			float u2dz= (texture) / pos3d_z1r-u1z; 
			float onez1 = 1/pos3d_z2r;
			float onedz2= 1/pos3d_z1r-onez1;

			u2dz  /=scr_y2r-scr_y1r;
			onedz2/=scr_y2r-scr_y1r;
#ifdef DETAIL_BENCH
			perf[x].elems_rendered++;
#endif

#ifdef PERPIXELFORWARD
			int skip_add = ofs_skip_start[scr_y2-1];
			int iskip_add  = ofs_skip_start[scr_y1];
			int sa=0;
			//skip_add += ofs_skip_start[scr_y2-1+skip_add]&65536;
			//iskip_add += ofs_skip_start[scr_y1+iskip_add]>>16;
			//scr_y2-=ofs_skip_start[scr_y2]>>16;
			//y+=ofs_skip_start[y+skip_add];
#endif
#ifdef HEIGHT_COLOR
			int height_color = 4095-mountain+viewpos.y;
#endif

			float mult = y+1-scr_y1r;
			float uz   = u1z   + u2dz  *mult;
			float onez = onez1 + onedz2*mult;
		
#ifdef PERPIXELFORWARD
			//#pragma unroll 2
			for( ;y<scr_y2;uz+=u2dz,onez+=onedz2)
#else
			//#pragma unroll 2
			for( ;y<scr_y2;++y,uz+=u2dz,onez+=onedz2)
#endif
			{
#ifdef PERPIXELFORWARD
				int skip=ofs_skip_start[y];//&65535;

				if (skip)	{	y+=skip;continue;	}
										  
				int skip_plus  = scr_y2-1-y;//+ skip_add;
				int skip_minus =-scr_y1+y  +iskip_add;
				ofs_skip_start[y]= skip_plus ;//+ ( skip_minus<<16 );
#endif
#ifdef NORMALCLIP
				if ( ((uint*)ofs_rgb_start) [y]!=0xff8844 ) continue; 				
#endif
#ifdef SHAREMEMCLIP
				int y5  = y>>5 ; 
				int y31 = 1<<(y&31);
				if( y_cache[y5] & y31 ) continue;
#endif
				uint u = min(max(float(uz/onez),float(texture)),float(tex-1.0));
				uint real_z = int(float(1/onez))&0xfffe;

#ifdef MEM_TEXTURE
				uint color16= tex1Dfetch(texture_slabs, send+u);
#else
				uint color16=send[u];
#endif

#ifdef HEIGHT_COLOR
				ushort colorpal=color16&0xff00;
				color16=min(max( (color16&0xff)*height_color>>12 , 0 ) , 255 );
				//if(color16<0)color16=0;
				//if(color16>255)color16=255;
				color16|=colorpal;
#endif
				((uint*)ofs_rgb_start) [y]  = color16+(real_z<<16);//+(real_z<<16);//depth;//send[uint(u)]
#ifdef PERPIXELFORWARD
				++y;
#endif				
#ifdef DETAIL_BENCH
				perf[x].pixels++;
#endif
#ifdef SHAREMEMCLIP
				y_cache[y5] |= y31;
#endif				
			}// pixel loop
		}//rle loop
//} // oversampling loop
};
	};
	/*------------------------------------------------------*/
};

hipArray* cu_array;
hipChannelFormatDesc channelDesc;

void create_cuda_1d_texture(char* h_data, int size)
{
	int d_size = ((size >> 8)+1)<<8;
	printf("d_size %d size %d \n",d_size,size);
	uint *d_octree;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_octree, d_size));
    CUDA_SAFE_CALL(hipMemcpy((void *)d_octree, (void *)h_data, size, hipMemcpyHostToDevice) );
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);

    // set texture parameters
    texture_slabs.addressMode[0] = hipAddressModeClamp;
    texture_slabs.addressMode[1] = hipAddressModeClamp;
    texture_slabs.addressMode[2] = hipAddressModeClamp;
    texture_slabs.filterMode = hipFilterModePoint;
    texture_slabs.normalized = false;    // access with normalized texture coordinates
    CUDA_SAFE_CALL(hipBindTexture(0, texture_slabs, d_octree, channelDesc) );
}
////////////////////////////////////////////////////////////////////////////////

hipArray* cu_array_pointermap;
hipChannelFormatDesc channelDesc_pointermap;

void create_cuda_2d_texture(uint* h_data, int width,int height)
{
	// Allocate CUDA array in device memory 
    channelDesc_pointermap = 
               hipCreateChannelDesc(32, 32, 0, 0,	
			   hipChannelFormatKindUnsigned); 
	    
    hipMallocArray(&cu_array_pointermap, &channelDesc_pointermap, width, height); 
 
    // Copy to device memory some data located at address h_data 
    // in host memory  
    hipMemcpyToArray(cu_array_pointermap, 0, 0, h_data, width*height*8, 
                      hipMemcpyHostToDevice); 
 
    // Set texture parameters 
    texture_pointermap.addressMode[0] = hipAddressModeClamp; 
    texture_pointermap.addressMode[1] = hipAddressModeClamp; 
    texture_pointermap.addressMode[2] = hipAddressModeClamp;
    texture_pointermap.filterMode     = hipFilterModePoint; 
    texture_pointermap.normalized     = false; 
 
    // Bind the array to the texture 
    hipBindTextureToArray(
		texture_pointermap, 
		cu_array_pointermap, 
		channelDesc_pointermap); 
	
	/*
	int d_size = (((size >> 8)+1)<<8);
	printf("d_size %d size %d \n",d_size,size);
	uint *d_data;

	channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);//<unsigned int>();//<unsigned int>();//
	CUDA_SAFE_CALL( hipMallocArray( &cu_array, &channelDesc, tex_w, tex_h )); 
	CUDA_SAFE_CALL( hipMemcpyToArray( cu_array, 0, 0, (void*)(texdata) , tex_w*tex_h*4, hipMemcpyHostToDevice));

	texture_array.addressMode[0] = hipAddressModeWrap;
	texture_array.addressMode[1] = hipAddressModeWrap;
    texture_array.addressMode[2] = hipAddressModeClamp;
	texture_array.filterMode = hipFilterModePoint;//hipFilterModeLinear;
	texture_array.normalized = false;    // access with normalized texture coordinates

	// Bind the array to the texture
	CUDA_SAFE_CALL( hipBindTextureToArray( texture_array, cu_array, channelDesc));
    CUDA_SAFE_CALL( hipDeviceSynchronize() );



    CUDA_SAFE_CALL(hipMalloc((void**) &d_octree, d_size));
    CUDA_SAFE_CALL(hipMemcpy((void *)d_octree, (void *)h_data, size, hipMemcpyHostToDevice) );
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

    // set texture parameters
    texture_array.addressMode[0] = hipAddressModeClamp;
    texture_array.addressMode[1] = hipAddressModeClamp;
    texture_array.filterMode = hipFilterModePoint;
    texture_array.normalized = false;    // access with normalized texture coordinates
    CUDA_SAFE_CALL(hipBindTexture(0, texture_array, d_octree, channelDesc) );
	*/
}
////////////////////////////////////////////////////////////////////////////////
// GL ERROR CHECK
int ChkGLError(char *file, int line)
{
	//return 0;
	return 0;
}
#define C_CHECK_GL_ERROR() ChkGLError(__FILE__, __LINE__)
////////////////////////////////////////////////////////////////////////////////
extern "C" void cuda_main_render2( int pbo_out, int width, int height,RayMap_GPU* raymap);
extern "C" void pboRegister(int pbo);
extern "C" void pboUnregister(int pbo);
intptr_t	cpu_to_gpu_delta=0;
////////////////////////////////////////////////////////////////////////////////
void gpu_memcpy(void* dst, void* src, int size)
{
	CUDA_SAFE_CALL( hipMemcpy( dst, src, size, hipMemcpyHostToDevice) );
	CUT_CHECK_ERROR("hipMemcpy hipMemcpyHostToDevice failed");
}
////////////////////////////////////////////////////////////////////////////////
void cpu_memcpy(void* dst, void* src, int size)
{
	CUDA_SAFE_CALL( hipMemcpy( dst, src, size, hipMemcpyDeviceToHost) );
	CUT_CHECK_ERROR("hipMemcpy hipMemcpyDeviceToHost failed");
}
////////////////////////////////////////////////////////////////////////////////
void* gpu_malloc(int size)
{
	void* ptr=0;	
	CUDA_SAFE_CALL( hipMalloc( (void**) &ptr, size ) );
	CUT_CHECK_ERROR("hipMalloc failed");
	if(ptr==0){printf("\ncudaMalloc %d MB: out of memory error\n",(size>>20));while(1);;}
	return ptr;
}
////////////////////////////////////////////////////////////////////////////////
__global__ void
cudaRender(
		   Render* render_local,
		   int maxrays, 
		   vec3f viewpos, 
		   vec3f viewrot, 
		   int res_x, 
		   int res_y,
		   ushort* skipmap_gpu
		  )
{
    extern __shared__ int sdata[];
   
    int x = ( blockIdx.y * 2 + blockIdx.x )* blockDim.x + threadIdx.x;
   
	//if(x&1)return;
    if (x>=maxrays) return;
    
    //render_local->render_line(x,(unsigned int*)&sdata[((x)&127)*31]);
    render_local->render_line
	(
		x,
		(unsigned int*)&sdata[((x)&(THREAD_COUNT-1))*(16300/(THREAD_COUNT*4))],//31
		viewpos,
		viewrot,
		res_x,
		res_y,
		skipmap_gpu+x*res_y
	);

	return;
}
////////////////////////////////////////////////////////////////////////////////
void cuda_main_render2( int pbo_out, int width, int height,RayMap_GPU* raymap)
{
	// int t0 = timeGetTime();

	if(pbo_out==0) return;

    static Render render;

	size_t render_len  = sizeof(Render);
	size_t skipmap_len = RAYS_CASTED*RENDER_SIZE*4;

	static Render* render_gpu  = (Render*)((uintptr_t)bmalloc(render_len)  + cpu_to_gpu_delta);
	static ushort* skipmap_gpu = (ushort*)((uintptr_t)bmalloc(skipmap_len) + cpu_to_gpu_delta);
    
    if((long)render_gpu==cpu_to_gpu_delta){ printf("render_gpu 0 \n");while(1);;}
    int lines_to_raycast = raymap->map_line_count;
    int thread_calls = ((raymap->map_line_count/2) | (THREAD_COUNT-1)) +1;
    if (lines_to_raycast>RAYS_CASTED ) lines_to_raycast=RAYS_CASTED;
    int* out_data;   
    CUDA_SAFE_CALL(cudaGLMapBufferObject( (void**)&out_data, pbo_out));   
	if(out_data==0) return;

	dim3 threads(THREAD_COUNT,1,1 );
    dim3 grid( 2 , thread_calls /(threads.x),1 );

    render.set_target( width, height, (int*) out_data);
  	render.set_raymap( raymap );

#ifdef DETAIL_BENCH
	for(int t=0;t<RAYS_CASTED;t++)
	{
		render.perf[t].elems_total=0;
		render.perf[t].elems_processed=0;
		render.perf[t].voxels_processed=0;
		render.perf[t].elems_rendered=0;
		render.perf[t].pixels=0;
	}
#endif
	
	gpu_memcpy(render_gpu, &render, sizeof(Render));
   
	// int t1 = timeGetTime();
	CUDA_SAFE_CALL( hipDeviceSynchronize() );

	//printf("before\n");
	//Sleep(10000);

	if(1)
	cudaRender<<< grid, threads, 16300 >>>
	(
		render_gpu,
		render.ray_map.map_line_count,
		render.ray_map.position,
		render.ray_map.rotation,
		render.res_x,
		render.res_y,
		skipmap_gpu
	);
	
	CUT_CHECK_ERROR("cudaRender failed");
//	CUT_CHECK_ERROR_GL();
	C_CHECK_GL_ERROR();

	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	// int t2 = timeGetTime();

#ifdef DETAIL_BENCH
	cpu_memcpy(&render.perf[0],&(render_gpu->perf[0]),  sizeof(Render::Perf)*RAYS_CASTED);
	Render::Perf p;
	p.elems_total=0;
	p.elems_processed=0;
	p.voxels_processed=0;
	p.elems_rendered=0;
	p.pixels=0;
	for(int t=0;t<RAYS_CASTED;t++)
	{
		p.elems_total+=render.perf[t].elems_total;
		p.elems_processed+=render.perf[t].elems_processed;
		p.voxels_processed+=render.perf[t].voxels_processed;
		p.elems_rendered+=render.perf[t].elems_rendered;
		p.pixels+=render.perf[t].pixels;
	}
	
	printf ("all %2.2fM proc %2.2fM vp %2.2fM ren %2.2fM pix %2.2fM ",
		float(p.elems_total)/(1000*1000),
		float(p.elems_processed)/(1000*1000),
		float(p.voxels_processed)/(1000*1000),
		float(p.elems_rendered)/(1000*1000),
		float(p.pixels)/(1000*1000));
#endif		
	//printf ("mem%d ren%d ",t1-t0,t2-t1);
    
    CUDA_SAFE_CALL(cudaGLUnmapBufferObject( pbo_out));
}
////////////////////////////////////////////////////////////////////////////////
void pboRegister(int pbo)
{
    // register this buffer object with CUDA
    CUDA_SAFE_CALL(cudaGLRegisterBufferObject(pbo));
	CUT_CHECK_ERROR("cudaGLRegisterBufferObject failed");
	C_CHECK_GL_ERROR();
}
////////////////////////////////////////////////////////////////////////////////
void pboUnregister(int pbo)
{
    // unregister this buffer object with CUDA
    CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(pbo));	
	CUT_CHECK_ERROR("cudaGLUnregisterBufferObject failed");
	C_CHECK_GL_ERROR();
}
////////////////////////////////////////////////////////////////////////////////
/*
__global__ void
cudaColorNodes(uint* nodebuf)
{
    int x = (blockIdx.x * blockDim.x + threadIdx.x);
    int y = (blockIdx.y * blockDim.y + threadIdx.y);

	ushort* node = (ushort*)(((uint*)nodebuf) [x+y*1024]);

	uint col_rgb=0xff8844;
	if(node)
	{
		ushort col=(ushort)*node;

		const int col_r[4]={130 ,255,255,155};
		const int col_g[4]={255 ,155,0  ,255};
		const int col_b[4]={130 ,0  ,0  ,0};						

		int col_o=(col>>8)&3;				
		int bright = col&255 ;

		int r_=(bright*col_r[col_o])>>8;
		int g_=(bright*col_g[col_o])>>8;
		int b_=(bright*col_b[col_o])>>8;

		col_rgb = r_+(g_<<8)+(b_<<16) ;
	}

	((uint*)nodebuf) [x+y*1024] = col_rgb;
}
*/
////////////////////////////////////////////////////////////////////////////////
