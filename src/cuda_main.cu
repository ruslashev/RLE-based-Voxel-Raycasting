#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#define IN_CUDA_ENV

#include "../inc/cutil_math.h"
#include "../inc/cutil.h"
#include "../inc/mathlib/matrixdefs.h"
#include <cuda_gl_interop.h>

#include "alloc.hh"
#include "ray_map.hh"
#include "rle4.hh"

texture<uint2, 2, hipReadModeElementType> texture_pointermap;
texture<unsigned short, 1, hipReadModeElementType> texture_slabs;

hipArray* cu_array;
hipChannelFormatDesc channelDesc;

hipArray* cu_array_pointermap;
hipChannelFormatDesc channelDesc_pointermap;

struct Render
{
	RayMap ray_map;
	int res_x, res_y;
	int* data_rgb;

	void set_target(int resolution_x, int resolution_y, int* data_rgb)
	{
		this->res_x = resolution_x;
		this->res_y = resolution_y;
		this->data_rgb = data_rgb;
		if (data_rgb == 0) {
			printf("data_rgb == 0\n");
			while (1) {}
		}
	}

	void set_raymap(RayMap* raymap)
	{
		memcpy(&ray_map, raymap, sizeof(RayMap));
	}

	inline __device__ float LineScale(vec3f input, vec3f center, float clip_max, float clip_min)
	{
		float scale_x = 1;
		float scale_y = 1;

		if (center.x > 1) scale_x = (1 - input.x) / (center.x - input.x);
		if (center.x < 0) scale_x = input.x / (input.x - center.x);
		if (center.y > clip_max) scale_y = (clip_max - input.y) / (center.y - input.y);
		if (center.y < clip_min) scale_y = (-clip_min + input.y) / (input.y - center.y);

		float scale = (scale_x < scale_y) ? scale_x : scale_y;

		return scale;
	}

	inline __device__ void ClipLine(vec3f& p1, vec3f& p2, float clip_max, float clip_min)
	{
		vec3f c1 = p1;
		vec3f c2 = p2;
		float scale;

		scale = LineScale(p1, p2, clip_max, clip_min);
		c2 = p1 + (p2 - p1) * scale;
		scale = LineScale(p2, p1, clip_max, clip_min);
		c1 = p2 + (p1 - p2) * scale;

		p1 = c1;
		p2 = c2;
	}

	inline __device__ vec3f MatMul(matrix44 m, vec3f v)
	{
		return make_float3(m.M11 * v.x + m.M21 * v.y + m.M31 * v.z + m.M41,
				m.M12 * v.x + m.M22 * v.y + m.M32 * v.z + m.M42,
				m.M13 * v.x + m.M23 * v.y + m.M33 * v.z + m.M43);
	}

	inline __device__ void vec3f_rot_y(float a, vec3f& v)
	{
		float xx = cos(-a) * v.x + sin(-a) * v.z;
		float zz = cos(-a) * v.z - sin(-a) * v.x;
		v.x = xx;
		v.z = zz;
	}

	inline __device__ void render_line(
			int x,
			unsigned int* y_cache,
			vec3f viewpos,
			vec3f viewrot,
			int res_x,
			int res_y,
			ushort* ofs_skip_start)
	{
		float res_x2 = res_x / 2;
		float res_y2 = res_y / 2;
		uint* ofs_rgb_start = (uint*)&data_rgb[x * res_y];
		uint* ofs_cache_start = ((uint*)ofs_skip_start) + x * res_y;

		ofs_cache_start[0] = 0;
		uint ofs_cache_count = 0;
		uint ofs_cache_depth = 0;

		float ml_ray_x;
		float ml_ray_z;
		vec3f ml_start2d;
		vec3f ml_end2d;
		vec3f ml_start3d;
		vec3f ml_end3d;
		bool ml_direction_y;

		{
			int rays[4];
			rays[0] = ray_map.res[0];
			rays[1] = ray_map.res[1] + rays[0];
			rays[2] = ray_map.res[2] + rays[1];
			rays[3] = ray_map.res[3] + rays[2];

			int quadrant = 0;
			if (x >= rays[2])
				quadrant = 3;
			else if (x >= rays[1])
				quadrant = 2;
			else if (x >= rays[0])
				quadrant = 1;

			float quadrant_ofs = x;
			if (quadrant >= 1)
				quadrant_ofs -= rays[quadrant - 1];

			float quadrant_num = ray_map.res[quadrant];
			float a = quadrant_ofs / quadrant_num;

			int j = quadrant;

			vec3f p1, p2, p1_3d, p2_3d;
			p1 = ray_map.vp;
			p2 = ray_map.p_no[j * 2] + (ray_map.p_no[j * 2 + 1] - ray_map.p_no[j * 2]) * a;

			ClipLine(p1, p2, ray_map.clip_max, ray_map.clip_min);

			matrix44 to3d = ray_map.to3d;
			vec3f p1m4 = p1 * 4.0;
			vec3f p2m4 = p2 * 4.0;
			p1_3d = MatMul(to3d, p1m4);
			p2_3d = MatMul(to3d, p2m4);

			vec3f delta = (p1_3d + p2_3d) * 0.5; // - origin
			delta.y = 0;
			delta = normalize(delta);
			vec3f_rot_y(viewrot.y, delta);

			ml_ray_x = delta.x;
			ml_ray_z = delta.z;
			ml_start2d = p1;
			ml_end2d = p2;
			ml_start3d = p1_3d;
			ml_end3d = p2_3d;
			ml_direction_y = 1 - (j >> 1);
		}

		// Initialize Render Vars
		int mip_lvl = 0;
		int y_clip_min = 0;
		int y_clip_max = res_y - 1;
		const int z_far = RAYS_DISTANCE;
		float dz = 1 << mip_lvl;
		int mapswitch = MIP_DISTANCE; // res_y2;

		// if (SCREEN_SIZE_X < res_y) mapswitch	= SCREEN_SIZE_X;

		// Initialize Rotation Vars
		float sin_x = sin(ray_map.rotation.x); // Rotation around x-axis
		float cos_x = cos(ray_map.rotation.x);
		float sin_y = sin(ray_map.rotation.y); // Rotation around y-axis
		float cos_y = cos(ray_map.rotation.y);

		// Initialize Raymap Vars
		float ray_x = ml_ray_x;
		float ray_z = ml_ray_z;
		bool vertical = ml_direction_y;

		// Reverse texturing ?
		bool reverse = false;
		if (vertical) if (ray_z <= 0) reverse = true;
		if (!vertical) if (ray_x <= 0) if (sin_x > 0) reverse = true;
		if (!vertical) if (ray_x > 0) if (sin_x < 0) reverse = true;

		float res_x2_mul_reverse = reverse ? -res_x2 : res_x2;

		if (vertical)
			res_x2_mul_reverse = -res_x2_mul_reverse;

		// Screenspace clipping
		int3 p1, p2;
		int p_add = reverse ? 1 : -2;
		p1.x = (int)((float)res_x * ml_start2d.x) + p_add;
		p1.y = (int)((float)res_y * ml_start2d.y) + p_add;
		p2.x = (int)((float)res_x * ml_end2d.x) - p_add;
		p2.y = (int)((float)res_y * ml_end2d.y) - p_add;
		if (p1.x < 0) p1.x = 0; if (p1.x >= res_x) p1.x = res_x - 1;
		if (p1.y < 0) p1.y = 0; if (p1.y >= res_y) p1.y = res_y - 1;
		if (p2.x < 0) p2.x = 0; if (p2.x >= res_x) p2.x = res_x - 1;
		if (p2.y < 0) p2.y = 0; if (p2.y >= res_y) p2.y = res_y - 1;

		if (p1.y == p2.y)
			return; // If removed -> Error ..!!?? Todo

		y_clip_min = res_x - 1 - p1.x;
		y_clip_max = res_x - 1 - p2.x;

		if (vertical) {
			y_clip_min = res_y - 1 - p1.y;
			y_clip_max = res_y - 1 - p2.y;
		}

		if (reverse) {
			y_clip_min = res_y - 1 - y_clip_min;
			y_clip_max = res_y - 1 - y_clip_max;
		}

		if (y_clip_min > y_clip_max) {
			int tmp = y_clip_min;
			y_clip_min = y_clip_max;
			y_clip_max = tmp;
		}

		if (y_clip_min >= y_clip_max)
			return;

		// Clear current rendered Line
		for (int n = y_clip_min; n <= y_clip_max; n++) {
			ofs_rgb_start[n] = 0xff8844;
#ifdef PERPIXELFORWARD
			ofs_skip_start[n] = 0;
#endif
		}

#ifdef SHAREMEMCLIP
		for (int n = 0; n < 31; n++)
			y_cache[n] = 0;
#endif

		float2 direction_rot;
		direction_rot.x = ray_x * cos_y + ray_z * sin_y;
		direction_rot.y = ray_x * sin_y - ray_z * cos_y;

		float2 delta = direction_rot, frac, fix, sign;

		fix.x = fix.y = -1;
		frac.x = viewpos.x - int(viewpos.x);
		frac.y = viewpos.z - int(viewpos.z);
		sign.x = sign.y = -1;

		// Signs & direction for frac
		if (delta.x >= 0) {
			fix.x = 0;
			sign.x = 1;
			frac.x = 1 - frac.x;
		}
		if (delta.y >= 0) {
			fix.y = 0;
			sign.y = 1;
			frac.y = 1 - frac.y;
		}

		// Gradients
		float2 grad0, grad1;
		grad0.y = delta.y / fabs(delta.x);
		grad0.x = sign.x;
		grad1.x = delta.x / fabs(delta.y);
		grad1.y = sign.y;

		// Intersections in x-,y- and z-plane
		float2 isect0, isect1;
		isect0.x = grad0.x * frac.x;
		isect0.y = grad0.y * frac.x;
		isect1.x = grad1.x * frac.y;
		isect1.y = grad1.y * frac.y;

		float grad_dist0 = sqrt(grad0.x * grad0.x + grad0.y * grad0.y);
		float grad_dist1 = sqrt(grad1.x * grad1.x + grad1.y * grad1.y);
		float dds_dist0 = sqrt(isect0.x * isect0.x + isect0.y * isect0.y);
		float dds_dist1 = sqrt(isect1.x * isect1.x + isect1.y * isect1.y);

		float2 pos_vxl_before = { 0, 0 };
		float dds_dist_before = 0;
		float2 pos_vxl = { 0, 0 };
		float dds_dist_now = 0;

		int index = 0, index_before = 0;

		// Main Render Loop
		int rle4_gridx = ray_map.map4_gpu[mip_lvl].sx;
		int rle4_gridz = ray_map.map4_gpu[mip_lvl].sz;

		float pos3d_z_add = sin_x;
		float pos3d_y_add = vertical ? cos_x : 0;
		pos3d_y_add *= res_x2_mul_reverse;

		uint* map_ptr = ray_map.map4_gpu[mip_lvl].map;
		ushort* slab_ptr = ray_map.map4_gpu[mip_lvl].slabs;

		float z = 0;

		float y_map_switch = viewpos.y;

		mapswitch = mapswitch * (0.25 * (4 - abs(viewrot.x)));

		uint tex_map_ofs = 0;

		while (true) {
			ofs_cache_depth++;

			while (z > mapswitch || y_map_switch > 512.0) {
				y_map_switch = y_map_switch * 0.5;

				if (mip_lvl < ray_map.nummaps - 1) {
					mip_lvl++;
					tex_map_ofs += rle4_gridz;
					rle4_gridx >>= 1;
					rle4_gridz >>= 1;
					map_ptr = ray_map.map4_gpu[mip_lvl].map;
					slab_ptr = ray_map.map4_gpu[mip_lvl].slabs;
				}

				grad0.x *= 2;
				grad0.y *= 2;
				grad1.x *= 2;
				grad1.y *= 2;
				grad_dist0 *= 2;
				grad_dist1 *= 2;
				mapswitch *= 2;
				dz *= 2;
			}

			z += dz;
			if (z > z_far)
				return;

			if ((y_clip_min >= y_clip_max))
				return;

			// DDA
			dds_dist_before = dds_dist_now;
			pos_vxl_before = pos_vxl;
			index_before = index;

			if (dds_dist1 < dds_dist0) {
				dds_dist_now = dds_dist1;
				index = 1;
				dds_dist1 += grad_dist1;
				pos_vxl = isect1;
				isect1.x += grad1.x;
				isect1.y += grad1.y;
			} else {
				dds_dist_now = dds_dist0;
				index = 0;
				pos_vxl = isect0;
				dds_dist0 += grad_dist0;
				isect0.x += grad0.x;
				isect0.y += grad0.y;
			}

			int fix_x = (1 - index_before) * fix.x, fix_z = (index_before)*fix.y;

			float dds_dist_delta = dds_dist_now - dds_dist_before;

			vec3f view_space;
			view_space.x = ray_x * dds_dist_before;
			view_space.z = ray_z * dds_dist_before;

			int voxel_x = (int(viewpos.x + pos_vxl_before.x) + fix_x);
			int voxel_z = (int(viewpos.z + pos_vxl_before.y) + fix_z);

#ifdef CLIPREGION
			if (voxel_x < 0) continue;
			if (voxel_z < 0) continue;
			if ((voxel_x >> mip_lvl) > rle4_gridx - 1) continue;
			if ((voxel_z >> mip_lvl) > rle4_gridz - 1) continue;
#endif

			int vx = (voxel_x >> mip_lvl) & (rle4_gridx - 1);
			int vz = (voxel_z >> mip_lvl) & (rle4_gridz - 1);

			float correct_x = ray_x * dds_dist_delta;
			float correct_z = ray_z * dds_dist_delta;

			float pos3d_z = cos_x * (view_space.z) + sin_x * viewpos.y;
			float pos3d_y = (vertical) ? cos_x * viewpos.y - sin_x * (view_space.z) : view_space.x;

			pos3d_y *= res_x2_mul_reverse;

			// Clip Top //if (pos3d_z>0)
			if (pos3d_z * res_y2 + pos3d_y <= pos3d_z * y_clip_min)
				continue;

			uint2 int64_ofs_rle = ((uint2*)map_ptr)[vx + vz * rle4_gridx];

			uint slab_offset = int64_ofs_rle.x;
			uint len_first = int64_ofs_rle.y;
			ushort slen = len_first;

			float pos3d_z1, pos3d_z2;
			int scr_y1;
			float pos3d_y1, pos3d_y2;
			int scr_y2;

			float corr_zz = cos_x * correct_z;
			float corr_yy = (vertical) ? -sin_x * correct_z : correct_x;

			corr_yy *= res_x2_mul_reverse;

			int sti_general = 0;
			int sti_skip = 0;
			ushort sti_;

			sti_ = len_first >> 16;

			ushort* slabs = slab_ptr + 2 + slab_offset;
			ushort* send = slabs + slen;

			float tex = 0;
			sti_general = 0;
			sti_skip = 0;

			uintptr_t slabs1 = (uintptr_t)slabs;

#pragma unroll 2
			for (; slabs < send; ++slabs) {
				if ((uintptr_t)slabs > slabs1)
					sti_ = *slabs;

				sti_skip = (sti_ >> 10);
				sti_general += (sti_ & 1023) << mip_lvl;
				if (sti_skip == 0)
					continue;

				int texture = tex;
				tex += sti_skip;
				sti_skip <<= mip_lvl;

				float sti_general_sti_skip = sti_general;
				sti_general += sti_skip;

				float correct_zz1 = pos3d_z;
				float correct_yy1 = pos3d_y;
				if (viewpos.y + sti_general_sti_skip >= 0) {
					correct_zz1 += corr_zz;
					correct_yy1 += corr_yy;
				}

				pos3d_z1 = correct_zz1 + pos3d_z_add * sti_general_sti_skip;
				if (pos3d_z1 <= 0)
					continue;
				pos3d_y1 = correct_yy1 + pos3d_y_add * sti_general_sti_skip;
				scr_y2 = res_y2 + pos3d_y1 / pos3d_z1;
				if (scr_y2 <= y_clip_min) {
					break;
				}

				float correct_zz2 = pos3d_z;
				float correct_yy2 = pos3d_y;
				if (viewpos.y + sti_general < 0) {
					correct_zz2 += corr_zz;
					correct_yy2 += corr_yy;
				}

				pos3d_z2 = correct_zz2 + pos3d_z_add * sti_general;
				if (pos3d_z2 <= 0)
					continue;
				pos3d_y2 = correct_yy2 + pos3d_y_add * sti_general;
				scr_y1 = res_y2 + pos3d_y2 / pos3d_z2 - 1;
				if (scr_y1 >= y_clip_max)
					continue;

				if (scr_y2 >= y_clip_max) {
					scr_y2 = y_clip_max;
#ifdef FLOATING_HORIZON
					y_clip_max = scr_y1;
#endif
				}
				if (scr_y1 <= y_clip_min) {
					scr_y1 = y_clip_min;
#ifdef FLOATING_HORIZON
					y_clip_min = scr_y2;
#endif
#ifdef SHAREMEMCLIP
	#ifdef XFLOATING_HORIZON
					while ((y_clip_max > y_clip_min)
							&& (y_cache[y_clip_min >> 5] & (1 << (y_clip_min & 31))))
						++y_clip_min;
	#endif
#endif
				}
				int y = scr_y1;

#ifdef PERPIXELFORWARD
				y += ofs_skip_start[y];
				// scr_y2-=ofs_skip_start[scr_y2-1]>>16;
	#ifdef XFLOATING_HORIZON
				y_clip_min += ofs_skip_start[y_clip_min];
	#endif
				if (y >= scr_y2)
					continue;
#endif

#ifdef SHAREMEMCLIP
				while ((y < scr_y2) && (y_cache[y >> 5] & (1 << (y & 31))))
					++y;
				if (y >= scr_y2)
					continue;
#endif
				float scr_y1r, pos3d_z2r, pos3d_y2r;
				float scr_y2r, pos3d_z1r, pos3d_y1r;

				pos3d_z1r = pos3d_z + pos3d_z_add * sti_general_sti_skip;
				pos3d_y1r = pos3d_y + pos3d_y_add * sti_general_sti_skip;

				pos3d_z2r = pos3d_z + pos3d_z_add * sti_general;
				pos3d_y2r = pos3d_y + pos3d_y_add * sti_general;

				scr_y2r = res_y2 + pos3d_y1r / pos3d_z1r;
				scr_y1r = res_y2 + pos3d_y2r / pos3d_z2r;

				float u1z = tex / pos3d_z2r;
				float u2dz = texture / pos3d_z1r - u1z;
				float onez1 = 1 / pos3d_z2r;
				float onedz2 = 1 / pos3d_z1r - onez1;

				u2dz /= scr_y2r - scr_y1r;
				onedz2 /= scr_y2r - scr_y1r;

#ifdef PERPIXELFORWARD
				int skip_add = ofs_skip_start[scr_y2 - 1];
				int iskip_add = ofs_skip_start[scr_y1];
				int sa = 0;
#endif

				float mult = y + 1 - scr_y1r;
				float uz = u1z + u2dz * mult;
				float onez = onez1 + onedz2 * mult;

#ifdef PERPIXELFORWARD
				for (; y < scr_y2; uz += u2dz, onez += onedz2)
#else
				for (; y < scr_y2; ++y, uz += u2dz, onez += onedz2)
#endif
				{
#ifdef PERPIXELFORWARD
					int skip = ofs_skip_start[y];

					if (skip) {
						y += skip;
						continue;
					}

					int skip_plus = scr_y2 - 1 - y; //+ skip_add;
					int skip_minus = -scr_y1 + y + iskip_add;
					ofs_skip_start[y] = skip_plus; //+ ( skip_minus<<16 );
#endif
#ifdef SHAREMEMCLIP
					int y5 = y >> 5;
					int y31 = 1 << (y & 31);
					if (y_cache[y5] & y31)
						continue;
#endif
					uint u = min(max(float(uz / onez), float(texture)), float(tex - 1.0));
					uint real_z = int(float(1 / onez)) & 0xfffe;

					uint color16 = send[u];

					((uint*)ofs_rgb_start)[y]
						= color16 + (real_z << 16); //+(real_z<<16);//depth;//send[uint(u)]
#ifdef PERPIXELFORWARD
					++y;
#endif
#ifdef SHAREMEMCLIP
					y_cache[y5] |= y31;
#endif
				} // pixel loop
			} // rle loop
		}
	}
};

void gpu_memcpy(void* dst, void* src, int size)
{
	CUDA_SAFE_CALL(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
	CUT_CHECK_ERROR("hipMemcpy hipMemcpyHostToDevice failed");
}

void* gpu_malloc(int size)
{
	void* ptr = 0;

	CUDA_SAFE_CALL(hipMalloc((void**)&ptr, size));
	CUT_CHECK_ERROR("hipMalloc failed");

	if (ptr == 0) {
		printf("\ncudaMalloc %d MB: out of memory error\n", (size >> 20));
		while (1) {}
	}

	return ptr;
}

__global__ void cuda_render(
		Render* render_local,
		int maxrays,
		vec3f viewpos,
		vec3f viewrot,
		int res_x,
		int res_y,
		ushort* skipmap_gpu)
{
	extern __shared__ int sdata[];

	int x = (blockIdx.y * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

	if (x >= maxrays)
		return;

	size_t idx = ((x) & (THREAD_COUNT - 1)) * (16300 / (THREAD_COUNT * 4));

	render_local->render_line(
			x,
			(unsigned int*)&sdata[idx],
			viewpos,
			viewrot,
			res_x,
			res_y,
			skipmap_gpu + x * res_y
		);

	return;
}

void cuda_main_render2(int pbo_out, int width, int height, RayMap* raymap)
{
	if (pbo_out == 0)
		return;

	static Render render;

	size_t render_len = sizeof(Render);
	size_t skipmap_len = RAYS_CASTED * RENDER_SIZE * 4;

	static Render* render_gpu = (Render*)((uintptr_t)bmalloc(render_len) + cpu_to_gpu_delta);
	static ushort* skipmap_gpu = (ushort*)((uintptr_t)bmalloc(skipmap_len) + cpu_to_gpu_delta);

	if ((long)render_gpu == cpu_to_gpu_delta) {
		printf("render_gpu == 0\n");
		while (1) {}
	}

	int lines_to_raycast = raymap->map_line_count;
	int thread_calls = ((raymap->map_line_count / 2) | (THREAD_COUNT - 1)) + 1;
	if (lines_to_raycast > RAYS_CASTED)
		lines_to_raycast = RAYS_CASTED;

	int* out_data;
	CUDA_SAFE_CALL(cudaGLMapBufferObject((void**)&out_data, pbo_out));
	if (out_data == 0)
		return;

	dim3 threads(THREAD_COUNT, 1, 1);
	dim3 grid(2, thread_calls / (threads.x), 1);

	render.set_target(width, height, (int*)out_data);
	render.set_raymap(raymap);

	gpu_memcpy(render_gpu, &render, sizeof(Render));

	CUDA_SAFE_CALL(hipDeviceSynchronize());

	cuda_render<<<grid, threads, 16300>>>(
			render_gpu,
			render.ray_map.map_line_count,
			render.ray_map.position,
			render.ray_map.rotation,
			render.res_x,
			render.res_y,
			skipmap_gpu);

	CUT_CHECK_ERROR("cudaRender failed");

	CUDA_SAFE_CALL(hipDeviceSynchronize());

	CUDA_SAFE_CALL(cudaGLUnmapBufferObject(pbo_out));
}

void cuda_pbo_register(int pbo)
{
	CUDA_SAFE_CALL(cudaGLRegisterBufferObject(pbo));
	CUT_CHECK_ERROR("cudaGLRegisterBufferObject failed");
}

void cuda_pbo_unregister(int pbo)
{
	CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(pbo));
	CUT_CHECK_ERROR("cudaGLUnregisterBufferObject failed");
}
